#include "hip/hip_runtime.h"
/* 
 * File:   main.cpp
 * Author: ian
 *
 * Created on June 26, 2011, 10:23 PM
 */

#include <mpi.h>
#include <omp.h>
#include <hip/hip_runtime.h>

float e;

// kernel
__global__ void sub1(float* fx, float* fy, float* fe) {
#define BLOCK (512)
  int t = threadIdx.x; // builtin
  int b = blockIdx.x; // builtin
  float e;
  __shared__ float se[BLOCK];
  __shared__ float sx[BLOCK];
  __shared__ float sy[BLOCK+2];
  // copy from device to processor memory
  sx[t] = fx[BLOCK*b+t];
  sy[t] = fy[BLOCK*b+t];
  if (t<2)
     sy[t+BLOCK] = fy[BLOCK*b+t+BLOCK];
  __syncthreads();

  // do computation
  sx[t] += ( sy[t+2] + sy[t] )*.5;
  e = sy[t+1] * sy[t+1];
  // copy to device memory
  fx[BLOCK*b+t] = sx[t];
  // reduction
  se[t] = e;
  __syncthreads();
  if (t<256) {
     se[t] += se[t+256];
     __syncthreads();
  }
  if (t<128) {
     se[t] += se[t+128];
     __syncthreads();
  }
  if (t<64) {
     se[t] += se[t+64];
     __syncthreads();
  }
  if (t<32) { // warp size
     se[t] += se[t+32];
     se[t] += se[t+16];
     se[t] += se[t+8];
     se[t] += se[t+4];
     se[t] += se[t+2];
     se[t] += se[t+1];
  }
  if (t==0)
     fe[b] = se[0];
}

int main(int argc, char *argv[]) {
  int n = 32;
  MPI_Init(&argc, &argv);
  int numproc, me;
  MPI_Comm_size(MPI_COMM_WORLD, &numproc);
  MPI_Comm_rank(MPI_COMM_WORLD, &me);
  int p_left = -1, p_right = -1;
  if (me > 0)
   p_left = me-1;
  if (me < numproc-1)
   p_right = me+1;
  int n_local0 = 1 + (me * (n-1)) / numproc;
  int n_local1 = 1 + ((me+1) * (n-1)) / numproc;
  // allocate only local part + ghost zone of the arrays x,y
  float *x, *y;
  x = new float[n_local1 - n_local0 + 2];
  y = new float[n_local1 - n_local0 + 2];
  x -= (n_local0 - 1);
  y -= (n_local0 - 1);

  // fill x, y

  // fill ghost zone
  MPI_Status s;
  if (p_left != -1)
    MPI_Send(&y[n_local0], 1, MPI_FLOAT, p_left,
      1, MPI_COMM_WORLD);
  if (p_right != -1) {
    MPI_Recv(&y[n_local1], 1, MPI_FLOAT, p_right,
      1, MPI_COMM_WORLD, &s);
    MPI_Send(&y[n_local1-1], 1, MPI_FLOAT, p_right,
      2, MPI_COMM_WORLD);
  }
  if (p_left != -1) 
    MPI_Recv(&y[n_local0-1], 1, MPI_FLOAT, p_left,
      2, MPI_COMM_WORLD, &s);
  

  e = 0;
  #pragma omp parallel
  {
  int p = omp_get_thread_num();
  int num = omp_get_num_threads();
  // pick GPU
  hipSetDevice(p);
  // allocate GPU memory
  float *fx, *fy, *fe;
  hipMalloc((void**)&fx, (n_local1-n_local0+2) * sizeof(float));
  hipMalloc((void**)&fy, (n_local1-n_local0+2) * sizeof(float));
  hipMalloc((void**)&fe, (n_local1-n_local0+2)/BLOCK * sizeof(float));
  float *de = new float[(n_local1-n_local0+2)/BLOCK];
  // copy to GPU memory
  hipMemcpy(fx+1, &x[n_local0],
   (n_local1-n_local0) * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(fy, &y[n_local0-1],
   (n_local1-n_local0+2) * sizeof(float), hipMemcpyHostToDevice);
  dim3 dimBlock(BLOCK, 1, 1);
  dim3 dimGrid((n_local1-n_local0+2)/BLOCK, 1, 1);

  int n0 = 1+((n_local1-n_local0)*p)/num;
  int n1 = 1+((n_local1-n_local0)*(p+1))/num;
  // call GPU
  sub1<<<dimGrid, dimBlock>>>(fx, fy, fe);
  // copy to host memory
  hipMemcpy(fx+1, &x[n0], (n1-n0) * sizeof(float),
   hipMemcpyDeviceToHost);
  hipMemcpy(fe, &de[n0-1], (n1-n0+2)/BLOCK * sizeof(float),
   hipMemcpyDeviceToHost);
  // release GPU memory
  hipFree(fe);
  hipFree(fy);
  hipFree(fx);
  float e_local = 0;
  for (int i=0; i<(n1-n0+2)/BLOCK; ++i)
   e_local += de[i];
  #pragma omp atomic
  e += e_local;
  delete[] de;
  }

  float e_local = e;
  MPI_Allreduce(&e_local, &e, 1, MPI_FLOAT, MPI_SUM, MPI_COMM_WORLD);

  // output x, e

  x += (n_local0 - 1);
  y += (n_local0 - 1);
  delete[] x, y;
  MPI_Finalize();
  return 0;
}
